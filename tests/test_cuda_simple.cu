#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */


#include "saiga/cuda/imageProcessing/NppiHelper.h"
//
#include "saiga/core/framework/framework.h"
#include "saiga/core/image/all.h"
#include "saiga/cuda/CudaInfo.h"
#include "saiga/cuda/imageProcessing/image.h"

#include "gtest/gtest.h"

#include "compare_numbers.h"

namespace Saiga
{
TEST(CudaSimple, Memcpy)
{
    int N = 10000;

    std::vector<int> h_data(N);

    for (auto& i : h_data)
    {
        i = Random::uniformInt(0, 100000);
    }

    size_t size = sizeof(int) * N;

    int* d_data;
    hipMalloc((void**)&d_data, size);
    int* d_data2;
    hipMalloc((void**)&d_data2, size);

    hipMemcpy(d_data, h_data.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_data2, d_data, size, hipMemcpyDeviceToDevice);

    std::vector<int> h_data2(N);
    hipMemcpy(h_data2.data(), d_data2, size, hipMemcpyDeviceToHost);

    EXPECT_EQ(h_data, h_data2);
}

__global__ static void addFive(int* data, int N)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= N) return;
    data[tid] = data[tid] + 5;
}


TEST(CudaSimple, AddFive)
{
    int N = 10000;
    std::vector<int> h_data(N);
    for (auto& i : h_data)
    {
        i = Random::uniformInt(0, 100000);
    }
    size_t size = sizeof(int) * N;
    int* d_data;
    hipMalloc((void**)&d_data, size);
    hipMemcpy(d_data, h_data.data(), size, hipMemcpyHostToDevice);


    addFive<<<iDivUp(N, 128), 128>>>(d_data, N);

    std::vector<int> h_data2(N);
    hipMemcpy(h_data2.data(), d_data, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i)
    {
        EXPECT_EQ(h_data2[i], h_data[i] + 5);
    }
}


}  // namespace Saiga

int main()
{
    Saiga::CUDA::initCUDA();
    Saiga::CUDA::printCUDAInfo();

    Saiga::initSaigaSampleNoWindow();
    testing::InitGoogleTest();

    return RUN_ALL_TESTS();
}
