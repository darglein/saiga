/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "stream.h"

#if !defined(_WIN32) && defined(SAIGA_USE_CUDA_TOOLKIT) && defined(SAIGA_CUDA_WITH_NVTOOLS)
#    include <nvToolsExtCudaRt.h>
#endif



namespace Saiga
{
namespace CUDA
{
CudaStream::CudaStream()
{
    hipStreamCreate(&stream);
}

CudaStream::~CudaStream()
{
    hipStreamDestroy(stream);
}

void CudaStream::waitForEvent(hipEvent_t event)
{
    hipStreamWaitEvent(stream, event, 0);
}

void CudaStream::synchronize()
{
    hipStreamSynchronize(stream);
}


hipStream_t CudaStream::legacyStream()
{
    return hipStreamLegacy;
}

hipStream_t CudaStream::perThreadStream()
{
    return hipStreamPerThread;
}

Saiga::CUDA::CudaStream::operator hipStream_t() const
{
    return stream;
}

void CudaStream::setName(const std::string& name)
{
#if !defined(_WIN32) && defined(SAIGA_USE_CUDA_TOOLKIT) && defined(SAIGA_CUDA_WITH_NVTOOLS)
    nvtxNameCudaStreamA(stream, name.c_str());
#else
    std::cerr << "CudaStream::setName only working if you enable SAIGA_CUDA_WITH_NVTOOLS in cmake" << std::endl;
#endif
}

}  // namespace CUDA
}  // namespace Saiga
