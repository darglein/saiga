#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/device_helper.h"
#include "saiga/cuda/imageProcessing/imageProcessing.h"

namespace Saiga
{
namespace CUDA
{
template <int BLOCK_W, int BLOCK_H, int ROWS_PER_THREAD = 1>
__global__ static void d_fill(ImageView<float> img, int h, float value)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int x = blockIdx.x * BLOCK_W + tx;
    int y = blockIdx.y * BLOCK_H + ty;

    if (x >= img.width) return;

    // process a fixed number of elements per thread to maximise instruction level parallelism
    for (int i = 0; i < ROWS_PER_THREAD; ++i, y += h)
    {
        if (y < img.height) img(y, x) = value;
    }
}

void fill(ImageView<float> img, float value)
{
    const int ROWS_PER_THREAD = 4;
    const int BLOCK_W         = 128;
    const int BLOCK_H         = 1;
    int w                     = img.width;
    int h                     = iDivUp(img.height, ROWS_PER_THREAD);
    dim3 blocks(iDivUp(w, BLOCK_W), iDivUp(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);
    d_fill<BLOCK_W, BLOCK_H, ROWS_PER_THREAD><<<blocks, threads>>>(img, h, value);
}



template <int BLOCK_W, int BLOCK_H, int ROWS_PER_THREAD = 1>
__global__ static void d_add(ImageView<float> img, int h, float value)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int x = blockIdx.x * BLOCK_W + tx;
    int y = blockIdx.y * BLOCK_H + ty;

    if (x >= img.width) return;

    // process a fixed number of elements per thread to maximise instruction level parallelism
    for (int i = 0; i < ROWS_PER_THREAD; ++i, y += h)
    {
        if (y < img.height) img(y, x) += value;
    }
}

void add(ImageView<float> img, float value)
{
    const int ROWS_PER_THREAD = 4;
    const int BLOCK_W         = 128;
    const int BLOCK_H         = 1;
    int w                     = img.width;
    int h                     = iDivUp(img.height, ROWS_PER_THREAD);
    dim3 blocks(iDivUp(w, BLOCK_W), iDivUp(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);
    d_add<BLOCK_W, BLOCK_H, ROWS_PER_THREAD><<<blocks, threads>>>(img, h, value);
}



template <int BLOCK_W, int BLOCK_H, int ROWS_PER_THREAD = 1>
__global__ static void d_mult(ImageView<float> img, int h, float value)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int x = blockIdx.x * BLOCK_W + tx;
    int y = blockIdx.y * BLOCK_H + ty;

    if (x >= img.width) return;

    // process a fixed number of elements per thread to maximise instruction level parallelism
    for (int i = 0; i < ROWS_PER_THREAD; ++i, y += h)
    {
        if (y < img.height) img(y, x) *= value;
    }
}

void mult(ImageView<float> img, float value)
{
    const int ROWS_PER_THREAD = 4;
    const int BLOCK_W         = 128;
    const int BLOCK_H         = 1;
    int w                     = img.width;
    int h                     = iDivUp(img.height, ROWS_PER_THREAD);
    dim3 blocks(iDivUp(w, BLOCK_W), iDivUp(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);
    d_mult<BLOCK_W, BLOCK_H, ROWS_PER_THREAD><<<blocks, threads>>>(img, h, value);
}


template <int BLOCK_W, int BLOCK_H, int ROWS_PER_THREAD = 1>
__global__ static void d_abs(ImageView<float> img, int h)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    int x = blockIdx.x * BLOCK_W + tx;
    int y = blockIdx.y * BLOCK_H + ty;

    if (x >= img.width) return;

    // process a fixed number of elements per thread to maximise instruction level parallelism
    for (int i = 0; i < ROWS_PER_THREAD; ++i, y += h)
    {
        if (y < img.height) img(y, x) = fabsf(img(y, x));
    }
}

void abs(ImageView<float> img)
{
    const int ROWS_PER_THREAD = 4;
    const int BLOCK_W         = 128;
    const int BLOCK_H         = 1;
    int w                     = img.width;
    int h                     = iDivUp(img.height, ROWS_PER_THREAD);
    dim3 blocks(iDivUp(w, BLOCK_W), iDivUp(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);
    d_abs<BLOCK_W, BLOCK_H, ROWS_PER_THREAD><<<blocks, threads>>>(img, h);
}



//#define USE_HARDWARE_INTER

#ifdef USE_HARDWARE_INTER
static texture<float, hipTextureType2D, hipReadModeElementType> floatTex;
#endif

template <int BLOCK_W, int BLOCK_H, int ROWS_PER_THREAD = 1>
__global__ static void d_scaleUp2Linear(ImageView<float> src, ImageView<float> dst, int h, double scale_x,
                                        double scale_y)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * BLOCK_W + tx;
    int y = blockIdx.y * BLOCK_H + ty;

    if (x >= dst.width) return;

#pragma unroll
    for (int i = 0; i < ROWS_PER_THREAD; ++i, y += h)
    {
        if (y < dst.height)
        {
#ifdef USE_HARDWARE_INTER
            // use hardware bil. interpolation
            float xf  = (float(x) + 0.5f) * scale_x;
            float yf  = (float(y) + 0.5f) * scale_y;
            dst(y, x) = tex2D(floatTex, xf, yf);
#else
            // software bil. interpolation
            float xf  = (float(x) + 0.5f) * scale_x - 0.5f;
            float yf  = (float(y) + 0.5f) * scale_y - 0.5f;
            dst(y, x) = src.inter(yf, xf);
#endif
        }
    }
}


void scaleUp2Linear(ImageView<float> src, ImageView<float> dst)
{
    SAIGA_ASSERT(src.width * 2 == dst.width && src.height * 2 == dst.height);

#ifdef USE_HARDWARE_INTER
    textureReference& floatTexRef = floatTex;
    hipChannelFormatDesc desc    = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    size_t offset;
    SAIGA_ASSERT(src.pitchBytes % 256 == 0);
    CHECK_CUDA_ERROR(hipBindTexture2D(&offset, &floatTexRef, src.data, &desc, src.width, src.height, src.pitchBytes));
    SAIGA_ASSERT(offset == 0);
    floatTexRef.addressMode[0] = hipAddressModeClamp;
    floatTexRef.addressMode[1] = hipAddressModeClamp;
    floatTexRef.filterMode     = hipFilterModeLinear;
    floatTexRef.normalized     = false;
#endif



    double inv_scale_x = (double)dst.width / src.width;
    double inv_scale_y = (double)dst.height / src.height;
    double scale_x = 1. / inv_scale_x, scale_y = 1. / inv_scale_y;


    const int ROWS_PER_THREAD = 4;
    const int BLOCK_W         = 128;
    const int BLOCK_H         = 1;
    int w                     = dst.width;
    int h                     = iDivUp(dst.height, ROWS_PER_THREAD);
    dim3 blocks(iDivUp(w, BLOCK_W), iDivUp(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);
    d_scaleUp2Linear<BLOCK_W, BLOCK_H, ROWS_PER_THREAD><<<blocks, threads>>>(src, dst, h, scale_x, scale_y);
}

}  // namespace CUDA
}  // namespace Saiga
