#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/core/util/statistics.h"
#include "saiga/cuda/device_helper.h"
#include "saiga/cuda/imageProcessing/imageProcessing.h"

namespace Saiga
{
namespace CUDA
{
#ifdef SAIGA_USE_EIGEN
thrust::device_vector<float> createGaussianBlurKernel(int radius, float sigma)
{
    SAIGA_ASSERT(radius <= SAIGA_MAX_CONVOLUTION_RADIUS && radius > 0);
#if 0
    const int ELEMENTS = radius * 2 + 1;
    thrust::host_vector<float> kernel(ELEMENTS);
    float kernelSum = 0.0f;
    float ivar2 = 1.0f/(2.0f*sigma*sigma);
    for (int j=-radius;j<=radius;j++) {
        kernel[j+radius] = (float)expf(-(double)j*j*ivar2);
        kernelSum += kernel[j+radius];
    }
    for (int j=-radius;j<=radius;j++)
        kernel[j+radius] /= kernelSum;
    return thrust::device_vector<float>(kernel);
#else
    auto kernel = gaussianBlurKernel1d<float>(radius, sigma);

    std::vector<float> f;
    for (int i = 0; i < kernel.rows(); ++i) f.push_back(kernel(i));
    return thrust::device_vector<float>(f);
#endif
}


void applyFilterSeparate(ImageView<float> src, ImageView<float> dst, ImageView<float> tmp, ArrayView<float> kernelRow,
                         ArrayView<float> kernelCol)
{
    convolveRow(src, tmp, kernelRow, kernelRow.size() / 2);
    convolveCol(tmp, dst, kernelCol, kernelCol.size() / 2);
}

void applyFilterSeparateSinglePass(ImageView<float> src, ImageView<float> dst, ArrayView<float> kernel)
{
    int radius = kernel.size() / 2;
    // inner 75 is the fastest for small kernels
    convolveSinglePassSeparateInner75(src, dst, kernel, radius);

#if 0
    return;
    if(radius <= 12 && src.cols < 32 && src.rows < 32)
    {
        convolveSinglePassSeparateInner75(src,dst,kernel,radius);
    }
    else if(radius <= 12)
    {
        convolveSinglePassSeparateInnerShuffle(src,dst,kernel,radius);

    }else
    {
        convolveSinglePassSeparateOuterHalo(src,dst,kernel,radius);
    }
#endif
}

#endif
}  // namespace CUDA
}  // namespace Saiga
