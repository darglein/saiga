#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2021 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/bitonicSort.h"
#include "saiga/cuda/cudaHelper.h"
#include "saiga/cuda/device_helper.h"
#include "saiga/cuda/pinned_vector.h"
#include "saiga/core/math/math.h"
#include "saiga/core/util/table.h"

#include "thrust/copy.h"
#include "thrust/scan.h"
#include "thrust/scatter.h"
#include "thrust/sort.h"

#include <iostream>
#include <vector>

//#define LECTURE

#ifdef LECTURE

static void radixSortHelper(thrust::device_vector<int>& d,
                            thrust::device_vector<int>& t, int bit)
{
}

static void radixSort(thrust::device_vector<int>& data)
{
    int N = data.size();
}

#else

template <bool one>
struct GetBitOp
{
    int k;
    GetBitOp(int k) : k(k) {}
    HD inline int operator()(int a) { return ((a >> k) & 1) == one; }
};

static void radixSortHelper(thrust::device_vector<int>& d, thrust::device_vector<int>& p, thrust::device_vector<int>& s,
                            thrust::device_vector<int>& t, int bit)
{
#if 0
    // Implementation with scan+scatter

    // Compute predicate array for 0-bits
    thrust::transform(d.begin(),d.end(),p.begin(),GetBitOp<false>(bit));

    // Scan over the predicate array and store it in s
    thrust::exclusive_scan(p.begin(),p.end(),s.begin(),0);

    // Write all 0-bit integers to the scanned positions
    // This writes only if the predicate also evaluates to true
    thrust::scatter_if(d.begin(),d.end(),s.begin(),p.begin(),t.begin());

    // Total number of 0 bits
//    int count = thrust::reduce(p.begin(),p.end());

    // Same with 1-bit integers, but use 'count' as the initial value in the scan
    thrust::transform(d.begin(),d.end(),p.begin(),GetBitOp<true>(bit));
    thrust::exclusive_scan(p.begin(),p.end(),s.begin(),count);
    thrust::scatter_if(d.begin(),d.end(),s.begin(),p.begin(),t.begin());
#else
    // Implementation with compact
    auto it = thrust::copy_if(d.begin(), d.end(), t.begin(), GetBitOp<false>(bit));
    thrust::copy_if(d.begin(), d.end(), it, GetBitOp<true>(bit));
#endif

    // The scan+scatter radix sort does not work inplace!
    thrust::copy(t.begin(), t.end(), d.begin());
}

static void radixSort(thrust::device_vector<int>& data)
{
    int N = data.size();

    // Temporary arrays
    thrust::device_vector<int> pred(N);
    thrust::device_vector<int> scan(N);
    thrust::device_vector<int> temp(N);

    // Sort from least to most significant bit
    for (int i = 0; i < 32; ++i) radixSortHelper(data, pred, scan, temp, i);
}

#endif
static void radixSortTest()
{
    int N   = 64 * 1024 * 1024;
    using T = int;
    Saiga::pinned_vector<T> h_data(N), res, res2;
    thrust::device_vector<T> d_data(N);

    // Initialize with random values
    for (auto& f : h_data)
    {
        f = abs(rand());
    }
    d_data = h_data;
    {
        std::cout << "Sorting " << N << " elements..." << std::endl;
        radixSort(d_data);
    }
    res = d_data;

    d_data = h_data;
    thrust::sort(d_data.begin(), d_data.end());
    res2 = d_data;

    SAIGA_ASSERT(res == res2);
    std::cout << "Success! All elements are in the correct order!" << std::endl;
}

int main(int argc, char* argv[])
{
    radixSortTest();
    std::cout << "Done." << std::endl;
}
